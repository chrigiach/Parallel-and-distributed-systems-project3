#include "hip/hip_runtime.h"
%%cu

/*
 * CUDA version:2(multiple threads per block, multiple work per thread)
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>


// ***Modulo for negative numbers*** (useful only for the device - GPU. It is used in Ising function)
__device__ int mod(int a, int b){
	int r = a % b;
    return r < 0 ? r + b : r;
}



// the mathematical formula that will be run on the GPU
__global__ void Ising(int* older, int* newer, int n, int w){
	
	//***inputs***
	// 'older' is the 2D square lattice that contains the initial info for the dipoles' moments
	// 'newer' is the 2D square lattice that will be used for the model's procedure(a simple exchange)
	// 'n' is the dimension of the square lattices
	//***output***
	// 'none': There is no output. The function does some calculations only.

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = i * n + j;
	int index = ind * w;
	int sum;
	// build the new phase of the material(or just create the formula that describes our model)
 	if ( i < n && j < n ){
		for(int y = 0 ; y < w ; y++){
			for(int x = 0 ; x < w ; x++){
				int r = i * w + y;
				int c = j * w + x;
				sum = older[r + n * c] + older[n * c + mod(r - 1, n)] + older[n * c + (r + 1) % n] + older[mod(c - 1, n) * n + r] + older[((c + 1) % n) * n + r];
				if(sum > 0){
					newer[r + n * c] = 1;
				} else {
					newer[r + n * c] = -1;
				}
			}
		}
	}
	__syncthreads();
}



// swap implementation
void swap(int **x, int **y){
	int *temp = *x;
	*x = *y;
	*y = temp;
}



int main(void){
	
	printf("\n");
	printf("Let's begin");
	printf("\n");
	
	//***___CPU variables___***
	int k = 10; // k iterations of the formula
	int n = 8; // the dimensions of the square 2D lattice
	int *G1, *G2; // the 2D square lattices stored in a form of an array
	
	// 1st 2D square lattice initialization
	G1 = (int *)malloc(n * n * sizeof(int));
	// filling the 2D square lattice(array) with -1 or 1
	for(int i = 0; i < n * n; i++){
			int random = (rand() % 2); // random numbers between 0 and 1
			if(random == 1){ // if random is 1 the G(i,j) is 1
				G1[i] = 1;
			} else { // if random is 0 the G(i,j) is -1
				G1[i] = -1;
			}
	}
	
	// 2nd 2D square lattice initialization
	G2 = (int *)malloc(n * n * sizeof(int));
	
	//***___GPU variables___***
	int *CUDAG1, *CUDAG2; // the 2D square lattices(arrays) that will exist in the GPU
	
  hipMalloc((void**)&CUDAG1, n * n * sizeof(int));

  hipMalloc((void**)&CUDAG2, n * n * sizeof(int));	
	



	printf("\n");
	printf("Let's start the procedure!!!");
	printf("\n");

	//***___CUDA parameters___***
	int m = 2; // for gpu blocks -> m*m blocks
	dim3 dimGrid(m, m); // grid size / number of blocks
	printf("m = : %d block_dimension \n", m);
	int t = 2; // t*tpb(t*t threads per block)
 	dim3 dimBlock(t, t); // Block size / number of threads
 	printf("t = : %d threads dimension\n", t);
	int w = 2; // w*w work per thread - how much blocks from the 2d square lattice are assigned to the thread
	printf("w = : %d work per thread\n", t);
	

	// initial state
	printf("The initial state of the ferromagnetic substance is: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			printf(" %d ",G1[i * n + j]);
		}
		printf("\n");
	}

	//***___start stopwatch___***
	clock_t begin = clock();
	
	//***___k iterations___***
	for(int i = 0 ; i < k ; i++){
		printf("*****____ iteration: %d ____***** \n", i);
		hipMemcpy(CUDAG1, G1, n*n*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(CUDAG2, G2, n*n*sizeof(int), hipMemcpyHostToDevice);
		Ising<<<dimGrid, dimBlock>>>(CUDAG1, CUDAG2, n, w);
		hipMemcpy(G1, CUDAG1, n*n*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(G2, CUDAG2, n*n*sizeof(int), hipMemcpyDeviceToHost);
		swap(&G1, &G2);
		for(int l = 0; l < n; l++){
			for(int j = 0; j < n; j++){
				printf(" %d ",G1[l * n + j]);
			}
			printf("\n");
		}
	}
	
	// stop stopwatch and print time
	clock_t end = clock();
	
	// The execution time
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	
	printf("\n");
	printf("End of the procedure!!!");
	printf("\n");
	
	// print the execution time
	printf("The time spent for execution was: %f \n", time_spent);

	
	// print the finished state of the moments(the G1 array, because it holds the results after the last swap)
 
	/*
	printf("The final state of the ferromagnetic substance is: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			printf(" %d ",G1[i * n + j]);
		}
		printf("\n");
	}
	*/
	// free the memory, do not need it anymore
	hipFree(CUDAG1);
	hipFree(CUDAG2);
	free(G1);
	free(G2);
	
	return 0;
}
