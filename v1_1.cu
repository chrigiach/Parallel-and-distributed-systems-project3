#include "hip/hip_runtime.h"
%%cu

/*
 * CUDA version:1.1(1 thread per block)
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>


// ***Modulo for negative numbers*** (useful only for the device - GPU. It is used in Ising function)
__device__ int mod(int a, int b){
	int r = a % b;
    return r < 0 ? r + b : r;
}



// the mathematical formula that will be run on the GPU
__global__ void Ising(int* older, int* newer, int n){

	//***inputs***
	// 'older' is the 2D square lattice that contains the initial info for the dipoles' moments
	// 'newer' is the 2D square lattice that will be used for the model's procedure(a simple exchange)
	// 'n' is the dimension of the square lattices
	//***output***
	// 'none': There is no output. The function does some calculations only.

	int i = blockIdx.x * blockDim.x;
	int j = blockIdx.y * blockDim.y;
	int sum;
	// build the new phase of the material(or just create the formula that describes our model)
	if ( i < n && j < n ){
		sum = older[n * i + j] + older[n * i + mod(j - 1, n)] + older[n * i + (j + 1) % n] + older[mod(i - 1, n) * n + j] + older[((i + 1) % n) * n + j];
		if(sum > 0){
			newer[n * i + j] = 1;
		} else {
			newer[n * i + j] = -1;
		}
	}
	__syncthreads();
}



// swap implementation
void swap(int **x, int **y){
	int *temp = *x;
	*x = *y;
	*y = temp;
}



int main(void){
	
	printf("\n");
	printf("Let's begin");
	printf("\n");
	
	//***___CPU variables___***
	int k = 10; // k iterations of the formula
	int n = 8; // the dimensions of the square 2D lattice
	int *G1, *G2; // the 2D square lattices stored in a form of an array
	
	// 1st 2D square lattice initialization
	G1 = (int *)malloc(n * n * sizeof(int));
	// filling the 2D square lattice(array) with -1 or 1
	for(int i = 0; i < n * n; i++){
			int random = (rand() % 2); // random numbers between 0 and 1
			if(random == 1){ // if random is 1 the G(i,j) is 1
				G1[i] = 1;
			} else { // if random is 0 the G(i,j) is -1
				G1[i] = -1;
			}
	}
	
	// 2nd 2D square lattice initialization
	G2 = (int *)malloc(n * n * sizeof(int));
	
	//***___GPU variables___***
	int *CUDAG1, *CUDAG2; // the 2D square lattices(arrays) that will exist in the GPU
	
  hipMalloc((void**)&CUDAG1, n * n * sizeof(int));

  hipMalloc((void**)&CUDAG2, n * n * sizeof(int));	
	



	printf("\n");
	printf("Let's start the procedure!!!");
	printf("\n");

	//***___CUDA parameters___***
	dim3 dimGrid(n, n); // grid size / number of blocks
	int thread_num = 1; // number of threads per block

	// initial state
	printf("The beginning state of the ferromagnetic substance is: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			printf(" %d ",G1[i * n + j]);
		}
		printf("\n");
	}

	//***___start stopwatch___***
	clock_t begin = clock();
	
	//***___k iterations___***
	for(int i = 0 ; i < k ; i++){
		printf("*****____ %d iteration: ____***** \n", i);
		hipMemcpy(CUDAG1, G1, n*n*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(CUDAG2, G2, n*n*sizeof(int), hipMemcpyHostToDevice);
		Ising<<<dimGrid, thread_num>>>(CUDAG1, CUDAG2, n);
		hipMemcpy(G1, CUDAG1, n*n*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(G2, CUDAG2, n*n*sizeof(int), hipMemcpyDeviceToHost);
		swap(&G1, &G2);
		for(int i = 0; i < n; i++){
			for(int j = 0; j < n; j++){
				printf(" %d ",G1[i * n + j]);
			}
			printf("\n");
		}
	}
	
	// stop stopwatch and print time
	clock_t end = clock();
	
	// The execution time
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	
	printf("\n");
	printf("End of the procedure!!!");
	printf("\n");
	
	// print the execution time
	printf("The time spent for execution was: %f \n", time_spent);

	
	// print the finished state of the moments(the G1 array, because it holds the results after the last swap)
 
	/*
	printf("The final state of the ferromagnetic substance is: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			printf(" %d ",G1[i * n + j]);
		}
		printf("\n");
	}
	*/
	// free the memory, do not need it anymore
	hipFree(CUDAG1);
	hipFree(CUDAG2);
	free(G1);
	free(G2);
	
	return 0;
}
